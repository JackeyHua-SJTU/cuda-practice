
#include <hip/hip_runtime.h>
#include <iostream>

static constexpr int SIZE = 256;

__global__ void simple_add(int *a, int *b, int *c) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int index = y * gridDim.x + x;
    if (index < SIZE) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int result[SIZE];
    int *dev_p;
    int count;
    
    hipGetDeviceCount(&count);
    printf("Total GPU count: %d\n", count);

    hipDeviceProp_t prop;
    // get the property of device 0
    hipGetDeviceProperties(&prop, 0);
    printf("Device 0 (%s) has warp size %d\n", prop.name, prop.warpSize);
    printf("    Major: %d, Minor: %d\n", prop.major, prop.minor);
    printf("    max thread per block: %d\n", prop.maxThreadsPerBlock);

    int a[SIZE], b[SIZE];
    for (int i = 0; i < SIZE; ++i) {
        a[i] = i + 1;
        b[i] = 2 * (i + 1);
    }
    int *var_a, *var_b;
    hipMalloc((void **)&var_a, SIZE * sizeof(int));
    hipMalloc((void **)&var_b, SIZE * sizeof(int));
    hipMemcpy(var_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(var_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&dev_p, SIZE * sizeof(int));
    dim3 grid(16, 16);  // 多维度必须这么写
    simple_add<<<grid, 1>>>(var_a, var_b, dev_p);
    hipMemcpy(result, dev_p, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_p);
    hipFree(var_a);
    hipFree(var_b);
    for (int i = 0; i < SIZE; ++i) {
        printf("answer[%d] is %d\n", i, result[i]);
    }

    return 0;
}