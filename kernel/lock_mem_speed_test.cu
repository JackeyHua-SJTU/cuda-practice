
#include <hip/hip_runtime.h>
#include <iostream>

/** @brief test the performance of copying malloc and cudaHostAlloc host memory to GPU 
 * 
 * Page-locked memory is much faster than noraml one because it is fixed in memory, and 
 * can not be swapped out to disk.
*/

static constexpr int SIZE = 30 * 1024 * 1024;

int main() {
    float elapsed_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int *host_mem = (int *)malloc(SIZE * sizeof(int));
    int *locked_host_mem;   // can not be paged and swapped into disk, always in memory
    hipHostAlloc((void **)&locked_host_mem, SIZE * sizeof(int), hipHostMallocDefault);

    int *dev_host_mem, *dev_locked_host_mem;
    hipMalloc((void **)&dev_host_mem, SIZE * sizeof(int));
    hipMalloc((void **)&dev_locked_host_mem, SIZE * sizeof(int));

    hipEventRecord(start);
    hipMemcpy(dev_host_mem, host_mem, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(host_mem, dev_host_mem, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(end);
    hipEventSynchronize(end);

    hipEventElapsedTime(&elapsed_time, start, end);
    printf("Normal malloc takes %.3f ms\n", elapsed_time);

    hipEventRecord(start);
    hipMemcpy(dev_locked_host_mem, locked_host_mem, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(locked_host_mem, dev_locked_host_mem, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(end);
    hipEventSynchronize(end);

    hipEventElapsedTime(&elapsed_time, start, end);
    printf("Locked malloc takes %.3f ms\n", elapsed_time);

    hipEventDestroy(start);
    hipEventDestroy(end);
    hipFree(dev_host_mem);
    hipFree(dev_locked_host_mem);
    hipHostFree(locked_host_mem);  // special free function
    free(host_mem);

    return 0;
}