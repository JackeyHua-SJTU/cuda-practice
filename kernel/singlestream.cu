
#include <hip/hip_runtime.h>
#include <iostream>

/** @brief Vector add with single stream 
 * 
 *  @note
 *  1. In stream and async data moving, we should use page-locked memory
 *  2. Use `cudaStreamSynchronize` to wait for certain stream to finish
 *  3. Blocking the computation. Deal with just a block in an iteration
*/

static constexpr int SIZE = 1024 * 1024;
static constexpr int TOTAL = 20 * SIZE;

__global__ void add(int *a, int *b, int *c) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (x < SIZE) {
        c[x] = a[x] + b[x];
        x += stride;
    }
}

int main() {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;

    hipHostAlloc((void **)&a, TOTAL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&b, TOTAL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&c, TOTAL * sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < TOTAL; ++i) {
        a[i] = i;
        b[i] = i;
    }

    hipMalloc((void **)&dev_a, SIZE * sizeof(int));
    hipMalloc((void **)&dev_b, SIZE * sizeof(int));
    hipMalloc((void **)&dev_c, SIZE * sizeof(int));

    hipStream_t stream;
    hipStreamCreate(&stream);
    
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, stream);
    
    for (int i = 0; i < TOTAL; i += SIZE) {
        hipMemcpyAsync(dev_a, a + i, SIZE * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, b + i, SIZE * sizeof(int), hipMemcpyHostToDevice, stream);
        add<<<32, 256, 0, stream>>>(dev_a, dev_b, dev_c);
        hipMemcpyAsync(c + i, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost, stream);
    }

    hipStreamSynchronize(stream);

    hipEventRecord(end, stream);
    hipEventSynchronize(end);

    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, end);
    
    printf("Total time: %.3f ms\n", elapsed_time);
    
    for (int i = 0; i < SIZE; ++i) {
        if (c[i] != i * 2) {
            printf("Error value of c array on index %d\n", i);
            exit(1);
        }
    }
    printf("Success: Every index has correct answer\n");

    hipEventDestroy(start);
    hipEventDestroy(end);
    hipStreamDestroy(stream);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(b);
    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}